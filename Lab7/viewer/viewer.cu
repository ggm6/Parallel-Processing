#include "hip/hip_runtime.h"
/* ACADEMIC INTEGRITY PLEDGE                                              */
/*                                                                        */
/* - I have not used source code obtained from another student nor        */
/*   any other unauthorized source, either modified or unmodified.        */
/*                                                                        */
/* - All source code and documentation used in my program is either       */
/*   my original work or was derived by me from the source code           */
/*   published in the textbook for this course or presented in            */
/*   class.                                                               */
/*                                                                        */
/* - I have not discussed coding details about this project with          */
/*   anyone other than my instructor. I understand that I may discuss     */
/*   the concepts of this program with other students and that another    */
/*   student may help me debug my program so long as neither of us        */
/*   writes anything during the discussion or modifies any computer       */
/*   file during the discussion.                                          */
/*                                                                        */
/* - I have violated neither the spirit nor letter of these restrictions. */
/*                                                                        */
/*                                                                        */
/*                                                                        */
/* Signed: Garrett McDonnell | Date:12/7/18                               */
/*                                                                        */
/*                                                                        */
/* 3460:677 CUDA Image Processing lab, Version 1.02, Fall 2016.           */


#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include "cpu_bitmap.h"
#include "bitmap_help.h"

__global__ void grayScale(unsigned char *device_arr);

__host__ void imgProc(unsigned char *map, int size, int width, int height);




int main(void) {
   char fname[50];
   FILE* infile;
   unsigned short ftype;
   tagBMFH bitHead;
   tagBMIH bitInfoHead;
   tagRGBQ *pRgb;

   printf("Please enter the .bmp file name: ");
   scanf("%s", fname);
   strcat(fname,".bmp");
   infile = fopen(fname, "rb");

   if (infile != NULL) {
      printf("File open successful.\n");
      fread(&ftype, 1, sizeof(unsigned short), infile);
      if (ftype != 0x4d42)
      {
         printf("File not .bmp format.\n");
         return 1;
      }
      fread(&bitHead, 1, sizeof(tagBMFH), infile);
      fread(&bitInfoHead, 1, sizeof(tagBMIH), infile);
   }
   else {
      printf("File open fail.\n");
      return 1;
   }

   if (bitInfoHead.biBitCount < 24) {
      long nPlateNum = long(pow(2, double(bitInfoHead.biBitCount)));
      pRgb = (tagRGBQ *)malloc(nPlateNum * sizeof(tagRGBQ));
      memset(pRgb, 0, nPlateNum * sizeof(tagRGBQ));
      int num = fread(pRgb, 4, nPlateNum, infile);
   }

   int width = bitInfoHead.biWidth;
   int height = bitInfoHead.biHeight;
   int l_width = 4 * ((width * bitInfoHead.biBitCount + 31) / 32);
   long nData = height * l_width;
   unsigned char *pColorData = (unsigned char *)malloc(nData);
   memset(pColorData, 0, nData);
   fread(pColorData, 1, nData, infile);

   fclose(infile);

   CPUBitmap dataOfBmp(width, height);
   unsigned char *map = dataOfBmp.get_ptr();

   if (bitInfoHead.biBitCount < 24) {
      int k, index = 0;
      if (bitInfoHead.biBitCount == 1) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 8;
               mixIndex = pColorData[k];
               if (j % 8 < 7) mixIndex = mixIndex << (7 - (j % 8));
               mixIndex = mixIndex >> 7;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 2) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 4;
               mixIndex = pColorData[k];
               if (j % 4 < 3) mixIndex = mixIndex << (6 - 2 * (j % 4));
               mixIndex = mixIndex >> 6;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 4) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 2;
               mixIndex = pColorData[k];
               if (j % 2 == 0) mixIndex = mixIndex << 4;
               mixIndex = mixIndex >> 4;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 8) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j;
               mixIndex = pColorData[k];
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 16) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j * 2;
               unsigned char shortTemp = pColorData[k + 1] << 8;
               mixIndex = pColorData[k] + shortTemp;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
   }
   else {
      int k, index = 0;
      for (int i = 0; i < height; i++)
         for (int j = 0; j < width; j++) {
            k = i * l_width + j * 3;
            map[index * 4 + 0] = pColorData[k + 2];
            map[index * 4 + 1] = pColorData[k + 1];
            map[index * 4 + 2] = pColorData[k];
            index++;
         }
   }

   imgProc(map, dataOfBmp.image_size(), width, height);
   dataOfBmp.display_and_exit();
   return 0;
}




__global__ void grayScale(unsigned char *device_arr){

   int pixel = (blockIdx.x + blockIdx.y * gridDim.x) * 4;
   
   unsigned char red = device_arr[pixel];
   
   unsigned char green = device_arr[pixel + 1];
   
   unsigned char blue = device_arr[pixel + 2];
   
   unsigned grayScale = 0.21 * red + 0.71 * green + 0.07 * blue;
   
   device_arr[pixel] = device_arr[pixel + 1] = device_arr[pixel + 2] = device_arr[pixel + 3] = grayScale;
   
}

__host__ void imgProc(unsigned char *map, int size, int width, int height) {

   unsigned char *device_arr;
   hipMalloc((void**) &device_arr, size);

   hipMemcpy(device_arr, map, size, hipMemcpyHostToDevice);

   dim3 dimGrid(width, height);
   dim3 dimBlock(1);
   
   grayScale<<<dimGrid, dimBlock>>>(device_arr);
   
   hipMemcpy(map, device_arr, size, hipMemcpyDeviceToHost);
   
   hipFree(device_arr);
   
   return;
}